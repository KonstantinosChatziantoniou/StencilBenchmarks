#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include "../headers/my_kernel_regs_32.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define TRUE 1
#define FALSE 0
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}




__global__ void my_32_8(float* g_input, float* g_output, // float* g_vsg, = 1
                        const int dimx, const int dimy, const int dimz)
{
    #define BDIMX 32 // tile (and threadblock) size in x
    #define BDIMY 32 // tile (and threadblock) size in y
    #define radius 8 // half of the order in space (k/2)

    __shared__ float tile[BDIMY+2*radius][BDIMX + 2*radius];
    //__shared__ float s_out[BDIMY][BDIMX][2*radius + 1];
    //float* z_data = all_z_data[threadIdx.x + threadIdx.y*BDIMX];
    int ix = blockIdx.x*blockDim.x + threadIdx.x  + radius;
    int iy = blockIdx.y*blockDim.y + threadIdx.y + radius;
    int stride = dimx*dimy; 
    int in_idx = iy*dimx + ix;                  // index for reading input
    //0int out_idx = in_idx + (radius-1)*stride;   // index for writing output

    int len = 2*radius + BDIMX; // Assuming BDIMX = BDIMY

    // int txr = threadIdx.x + radius;              
    // int tyr = threadIdx.y + radius;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int txr = threadIdx.x + radius;              // thread’s x-index into corresponding shared memory tile (adjusted for halos)
    int tyr = threadIdx.y + radius;
    int tidx = threadIdx.x;//(threadIdx.x + 32*threadIdx.y)%32;
    int tidy = threadIdx.y;//(threadIdx.x + 32*threadIdx.y)/32;
    int tidix = blockIdx.x*blockDim.x + tidx;
    int tidiy = blockIdx.y*blockDim.y + tidy;
    //int stride = dimx*dimy; 
    int tid_in_idx = tidiy*dimx + tidix;        
    // Init out shared array
    // for(int i = 0; i < (2*radius+1); i++){
    //     s_out[ty][tx][i] = 0;   
    // }
    float behind8 = 0;
    float behind7 = 0;
    float behind6 = 0;
    float behind5 = 0;
    float behind4 = 0;
    float behind3 = 0;
    float behind2 = 0;
    float behind1 = 0;
    float current = 0;
    float infront1 = 0;
    float infront2 = 0;
    float infront3 = 0;
    float infront4 = 0;
    float infront5 = 0;
    float infront6 = 0;
    float infront7 = 0;
    float infront8 = 0;
    tid_in_idx += (radius)*stride;
    for(int z = radius; z < dimz - radius; z++){
        // read tile with halo
        __syncthreads();
        for(int i = 0; i < len; i += 32){
            for(int j = 0; j < len; j += 32){
                if(tidx + i < len && tidy + j < len){
                    tile[j+tidy][i+tidx] = g_input[tid_in_idx + i + j*dimx];
                    //printf("%d %d %d %d\n",tx,ty, j+tidy, i+tidx);
                }
            }
        }

        __syncthreads();
        current += 3*tile[tyr][txr];
        // for(int i = 1; i <= radius; i++){
        //     current += (tile[tyr][txr+i]
        //     + tile[tyr+i][txr] + tile[tyr-i][txr]
        //     + tile[tyr][txr-i]);
        //     // s_out[ty][tx][4+i] += tile[tyr][txr];
        //     // s_out[ty][tx][i-1] += tile[tyr][txr];
        // }
        current += (tile[tyr][txr+1]
            + tile[tyr+1][txr] + tile[tyr-1][txr]
            + tile[tyr][txr-1]);
        current += (tile[tyr][txr+2]
            + tile[tyr+2][txr] + tile[tyr-2][txr]
            + tile[tyr][txr-2]);
        current += (tile[tyr][txr+3]
            + tile[tyr+3][txr] + tile[tyr-3][txr]
            + tile[tyr][txr-3]);
        current += (tile[tyr][txr+4]
            + tile[tyr+4][txr] + tile[tyr-4][txr]
            + tile[tyr][txr-4]);
        current += (tile[tyr][txr+5]
            + tile[tyr+5][txr] + tile[tyr-5][txr]
            + tile[tyr][txr-5]);
        current += (tile[tyr][txr+6]
            + tile[tyr+6][txr] + tile[tyr-6][txr]
            + tile[tyr][txr-6]);
        current += (tile[tyr][txr+7]
            + tile[tyr+7][txr] + tile[tyr-7][txr]
            + tile[tyr][txr-7]);
        current += (tile[tyr][txr+8]
            + tile[tyr+8][txr] + tile[tyr-8][txr]
            + tile[tyr][txr-8]);
        float temp = tile[tyr][txr];
        behind8 += temp;
        behind7 += temp;
        behind6 += temp;
        behind5 += temp;
        behind4 += temp;
        behind3 += temp;
        behind2 += temp;
        behind1 += temp;
        infront1 +=  temp;
        infront2 +=  temp;
        infront3 +=  temp;
        infront4 +=  temp;
        infront5 +=  temp;
        infront6 +=  temp;
        infront7 +=  temp;
        infront8 +=  temp;
        if(z > 2*radius){
            g_output[in_idx] = behind8;
        }
        // for(int i = 0; i < 2*radius; i++){
        //     s_out[ty][tx][i] = s_out[ty][tx][1+i];
        // }
        behind8 = behind7;
        behind7 = behind6;
        behind6 = behind5;
        behind5 = behind4;
        behind4 = behind3;
        behind3 = behind2;
        behind2 = behind1;
        behind1 = current;
        current = infront1;
        infront1 = infront2;
        infront2 = infront3;
        infront3 = infront4;
        infront4 = infront5;
        infront5 = infront6;
        infront6 = infront7;
        infront7 = infront8;
        infront8 = 0;  
        in_idx += stride;
        tid_in_idx += stride;
    }

    // for(int i = 1; i < radius; i++){
    //     g_output[in_idx + (dimz-radius+i-1)*stride] =  s_out[ty][tx][i-1];
    // }
    // in_idx += stride;
    g_output[in_idx] = behind8;
    in_idx += stride;
    g_output[in_idx] = behind7;
    in_idx += stride;
    g_output[in_idx] = behind6;
    in_idx += stride;
    g_output[in_idx] = behind5;
    in_idx += stride;
    g_output[in_idx] = behind4;
    in_idx += stride;
    g_output[in_idx] = behind3;
    in_idx += stride;
    g_output[in_idx] = behind2;
    in_idx += stride;
    g_output[in_idx] = behind1;
    
}








float* myStencil32_8(float* data, int dimx, int dimy, int dimz){
    float* out = (float*)malloc(dimx*dimy*dimz*sizeof(float));
    for(int i = 0; i < dimx*dimy*dimz; i++){
        out[i] = 0;
    }
    float *dev_data, *dev_out;
    gpuErrchk(hipMalloc((void**)&dev_data, dimx*dimy*dimz*sizeof(float)))
    gpuErrchk(hipMalloc((void**)&dev_out, dimx*dimy*dimz*sizeof(float)))

    gpuErrchk(hipMemcpy(dev_data, data, dimx*dimy*dimz*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_out, out, dimx*dimy*dimz*sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks((dimx-2*8)/32, (dimy-2*8)/32, 1);
    dim3 threads(32, 32, 1);
    my_32_8<<<blocks,threads>>>(dev_data, dev_out, dimx,dimy, dimz);
    gpuErrchk(hipMemcpy(out, dev_out, dimx*dimy*dimz*sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree((void**)dev_data));
    gpuErrchk(hipFree((void**)dev_out));
    return out;
}